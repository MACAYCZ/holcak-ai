#include "hip/hip_runtime.h"
#include "trainer.cuh"

namespace HAI
{
	Trainer::Trainer(Network network, std::size_t batch, std::size_t threads)
		: network(network)
		, batch(batch)
		, threads(threads)
	{
		hipMallocManaged(&this->layers, network.size * sizeof(Layer));
		for (std::size_t i = 0; i < network.size; i++) {
			hipMallocManaged(&this->layers[i].weighted, batch * network.layers[i].outputs * sizeof(double));
			hipMallocManaged(&this->layers[i].activated, batch * network.layers[i].outputs * sizeof(double));
		}
	}

	Trainer::~Trainer()
	{
		hipFree(&this->layers);
	}

	__global__ void Calculate()
	{
		// TODO!
	}

	__global__ void Update()
	{
		// TODO!
	}

	void Trainer::Step(const double* inputs, const double* outputs)
	{
		// TODO!
	}
}
